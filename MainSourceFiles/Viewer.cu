#include "hip/hip_runtime.h"
/*
nvcc Viewer.cu -o Viewer.exe -lglut -lGL -lGLU -lm
nvcc Viewer.cu -o Viewer.exe -lglut -lGL -lGLU -lm --use_fast_math
*/

#include "../CommonCompileFiles/binaryStarCommonIncludes.h"
#include "../CommonCompileFiles/binaryStarCommonDefines.h"
#include "../CommonCompileFiles/binaryStarCommonGlobals.h"
#include "../CommonCompileFiles/binaryStarCommonFunctions.h"
#include "../CommonCompileFiles/binaryStarCommonRunGlobals.h"
#include "../CommonCompileFiles/binaryStarCommonRunFunctions.h"

//Time to add on to the run. Readin from the comand line.
float ContinueRunTime;

void openAndReadFiles()
{
	ifstream data;
	string name;
	
	//Opening the positions and velosity file to dump stuff to make movies out of. Need to move to the end of the file.
	PosAndVelFile = fopen("PosAndVel", "rb+");
	if(PosAndVelFile == NULL)
	{
		printf("\n\n The PosAndVel file does not exist\n\n");
		exit(0);
	}
	//fseek(PosAndVelFile,0,SEEK_END);
	
	//Reading in the run parameters
	data.open("RunParameters");
	if(data.is_open() == 1)
	{
		getline(data,name,'=');
		data >> SystemLengthConverterToKilometers;
		
		getline(data,name,'=');
		data >> SystemMassConverterToKilograms;
		
		getline(data,name,'=');
		data >> SystemTimeConverterToSeconds;
		
		getline(data,name,'=');
		data >> NumberElementsStar1;
		
		getline(data,name,'=');
		data >> NumberElementsStar2;
		
		getline(data,name,'=');
		data >> CoreCorePushBackReduction;
		
		getline(data,name,'=');
		data >> CorePlasmaPushBackReduction;
		
		getline(data,name,'=');
		data >> PlasmaPlasmaPushBackReduction;
		
		getline(data,name,'=');
		data >> Dt;
		
		getline(data,name,'=');
		data >> ZoomFactor;
		
		getline(data,name,'=');
		data >> PrintRate;
	}
	else
	{
		printf("\nTSU Error could not open RunParameters file\n");
		exit(0);
	}
	data.close();
	NumberElements = NumberElementsStar1 + NumberElementsStar2;
}

void drawPictureViewer()
{	
	double diameterSun;
	double drawUnit;
	
	diameterSun = DIAMETER_SUN/SystemLengthConverterToKilometers;
	drawUnit = 1.0/(diameterSun/ZoomFactor);
	
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
	
	//Drawing the cores spheres
	glPushMatrix();
		glTranslatef(drawUnit*(PosCPU[0].x - CenterOfView.x), drawUnit*(PosCPU[0].y - CenterOfView.y), drawUnit*(PosCPU[0].z - CenterOfView.z));
		glColor3d(1.0,0.0,0.0);
		glutSolidSphere(drawUnit*diameterSun*0.2/2.0,20,20);  // force.w holds the diameter of an element
	glPopMatrix();
	
	glPushMatrix();
		glTranslatef(drawUnit*(PosCPU[NumberElementsStar1].x - CenterOfView.x), drawUnit*(PosCPU[NumberElementsStar1].y - CenterOfView.y), drawUnit*(PosCPU[NumberElementsStar1].z - CenterOfView.z));
		glColor3d(0.0,0.0,1.0);
		glutSolidSphere(drawUnit*diameterSun*0.2/2.0,20,20);
	glPopMatrix();
	
	//Drawing all the elements as points
	glBegin(GL_POINTS);
		glPointSize(5.0);
		glColor3d(1.0,1.0,0.0);
 		for(int i = 0; i < NumberElementsStar1; i++)
		{
			glVertex3f(drawUnit*(PosCPU[i].x - CenterOfView.x), drawUnit*(PosCPU[i].y - CenterOfView.y), drawUnit*(PosCPU[i].z - CenterOfView.z));
		}
		glColor3d(1.0,0.6,0.0);
		for(int i = NumberElementsStar1; i < NumberElements; i++)
		{
			glVertex3f(drawUnit*(PosCPU[i].x - CenterOfView.x), drawUnit*(PosCPU[i].y - CenterOfView.y), drawUnit*(PosCPU[i].z - CenterOfView.z));
		}
	glEnd();
	
	glutSwapBuffers();
}

void control()
{	
	float time;
	
	// Reading in the build parameters.
	printf("\n Reading and setting the run parameters.\n");
	openAndReadFiles();
	
	// Allocating memory for CPU and GPU.
	printf("\n Allocating memory on the GPU and CPU and opening positions and velocities file.\n");
	allocateCPUMemory();
	
	float temp = -10.0;
	int stop = 0;
	while(stop != 1)
	{
		fread(&time, sizeof(float), 1, PosAndVelFile);
		if(temp - time == 0.0) 
		{
			stop = 1;
		}
		temp = time;
		fread(PosCPU, sizeof(float4), NumberElements, PosAndVelFile);
		fread(VelCPU, sizeof(float4), NumberElements, PosAndVelFile);
		printf("\n time =%f", time);
		drawPictureViewer();
	}
	printf("\n The run has finished successfully \n\n");
	//while(1);
	
	// Freeing memory. 	
	printf("\n Cleaning up the run.\n");
	//cleanUp(gPUsUsed);
	fclose(PosAndVelFile);

	exit(0);
}

int main(int argc, char** argv)
{ 
	glutInit(&argc,argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_DEPTH | GLUT_RGB);
	glutInitWindowSize(XWindowSize,YWindowSize);
	glutInitWindowPosition(0,0);
	glutCreateWindow("Creating Stars");
	
	glutReshapeFunc(reshape);
	
	init();
	
	glShadeModel(GL_SMOOTH);
	glClearColor(0.0, 0.0, 0.0, 0.0);
	
	glutDisplayFunc(Display);
	glutReshapeFunc(reshape);
	glutIdleFunc(control);
	glutMainLoop();
	return 0;
}

